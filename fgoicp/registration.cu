#include "hip/hip_runtime.h"
#include "registration.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/system/cuda/execution_policy.h>


namespace icp
{
    __global__ void kernComputeClosestError(int N, glm::mat3 R, glm::vec3 t, const Point3D *d_pcs, const FlattenedKDTree* d_fkdt, float* d_errors)
    {
        int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (index >= N) { return; }

        Point3D source_point = d_pcs[index];
        Point3D query_point = R * source_point + t;

        size_t nearest_index = 0;
        float distance_squared = M_INF;
        d_fkdt->find_nearest_neighbor(query_point, distance_squared, nearest_index);

        d_errors[index] = distance_squared;
    }

    __global__ void kernComputeBounds(int N, RotNode rnode, TransNode tnode, bool fix_rot, const Point3D* d_pcs, const FlattenedKDTree* d_fkdt, float* d_rot_ub_trans_ub, float* d_rot_ub_trans_lb)
    {
        int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (index >= N) { return; }

        Point3D source_point = d_pcs[index];
        float trans_uncertain_radius = M_SQRT3 * tnode.span;
        Point3D query_point = rnode.q.R * source_point + tnode.t;

        float rot_uncertain_radius;
        if (!fix_rot)
        {
            float radius = source_point.x * source_point.x +
                source_point.y * source_point.y +
                source_point.z * source_point.z;
            float half_angle = rnode.span * M_SQRT3 * M_PI / 2.0f;  // TODO: Need examination, since we are using quaternions
            rot_uncertain_radius = 2.0f * radius * sin(half_angle);
        }

        size_t nearest_index = 0;
        float distance_squared = M_INF;
        d_fkdt->find_nearest_neighbor(query_point, distance_squared, nearest_index);

        float distance = sqrt(distance_squared);
        if (!fix_rot)
        {
            distance -= rot_uncertain_radius;
        }

        d_rot_ub_trans_ub[index] = distance > 0.0f ? distance * distance : 0.0f;


        float rot_ub_trans_lb = distance - trans_uncertain_radius;
        rot_ub_trans_lb = rot_ub_trans_lb > 0.0f ? rot_ub_trans_lb * rot_ub_trans_lb : 0.0f;
        d_rot_ub_trans_lb[index] = rot_ub_trans_lb;
    }
    
    float Registration::compute_sse_error(glm::mat3 R, glm::vec3 t) const
    {
        float* dev_errors;
        hipMalloc((void**)&dev_errors, sizeof(float) * ns);

        size_t block_size = 32;
        dim3 threads_per_block(block_size);
        dim3 blocks_per_grid((ns + block_size - 1) / block_size);
        kernComputeClosestError <<<blocks_per_grid, threads_per_block>>> (
            ns, R, t,
            thrust::raw_pointer_cast(d_pcs.data()),
            d_fkdt,
            dev_errors);
        hipDeviceSynchronize();
        cudaCheckError("Kernel launch");

        // Sum up the squared errors with thrust::reduce
        thrust::device_ptr<float> dev_errors_ptr(dev_errors);
        float sse_error = thrust::reduce(dev_errors_ptr, dev_errors_ptr + ns, 0.0f, thrust::plus<float>());
        cudaCheckError("thrust::reduce");

        hipFree(dev_errors);

        return sse_error;
    }

    Registration::BoundsResult_t Registration::compute_sse_error(RotNode &rnode, std::vector<TransNode> &tnodes, bool fix_rot, StreamPool& stream_pool) const
    {
        size_t num_transforms = tnodes.size();
        std::vector<float> sse_rot_ub_trans_ub(num_transforms);
        std::vector<float> sse_rot_ub_trans_lb(num_transforms);

        // Allocate memory on the device for the errors for each (R, t) pair
        float* d_rot_ub_trans_ub;
        float* d_rot_ub_trans_lb;
        hipMalloc((void**)&d_rot_ub_trans_ub, sizeof(float) * ns * num_transforms);
        hipMalloc((void**)&d_rot_ub_trans_lb, sizeof(float) * ns * num_transforms);

        thrust::device_ptr<float> d_thrust_rot_ub_trans_ub(d_rot_ub_trans_ub);
        thrust::device_ptr<float> d_thrust_rot_ub_trans_lb(d_rot_ub_trans_lb);

        // Kernel launching parameters
        size_t block_size = 32;
        dim3 threads_per_block(block_size);
        dim3 blocks_per_grid((ns + block_size - 1) / block_size);

        // Launch kernel for each (R, t) pair on separate streams
        for (size_t i = 0; i < num_transforms; ++i) {
            // Get the appropriate stream from the stream pool
            hipStream_t stream = stream_pool.getStream(i);

            // Launch the kernel with each (R, t) on a different stream
            kernComputeBounds <<<blocks_per_grid, threads_per_block, 0, stream>>> (
                ns, rnode, tnodes[i], fix_rot,
                thrust::raw_pointer_cast(d_pcs.data()),
                d_fkdt,
                d_rot_ub_trans_ub + i * ns,
                d_rot_ub_trans_lb + i * ns);
        }

        // Reduce the lower/upper bounds for each pair
        for (size_t i = 0; i < num_transforms; ++i) {
            // Thrust reduce launching parameters
            auto thrust_policy = thrust::cuda::par.on(stream_pool.getStream(i));

            sse_rot_ub_trans_ub[i] = thrust::reduce(
                thrust_policy,
                d_thrust_rot_ub_trans_ub + i * ns,
                d_thrust_rot_ub_trans_ub + (i + 1) * ns,
                0.0f,
                thrust::plus<float>()
            );

            sse_rot_ub_trans_lb[i] = thrust::reduce(
                thrust_policy,
                d_thrust_rot_ub_trans_lb + i * ns,
                d_thrust_rot_ub_trans_lb + (i + 1) * ns,
                0.0f,
                thrust::plus<float>()
            );
        }

        hipDeviceSynchronize();

        // Free the device memory
        hipFree(d_rot_ub_trans_ub);
        hipFree(d_rot_ub_trans_lb);

        return { sse_rot_ub_trans_lb, sse_rot_ub_trans_ub };
    }


    //============================================
    //            Flattened k-d tree
    //============================================
    
    FlattenedKDTree::FlattenedKDTree(const KDTree& kdt, const PointCloud& pct) :
        h_vAcc{kdt.vAcc_},
        h_pct{pct.begin(), pct.end()}
    {
        // Convert KDTree to array on the host
        size_t currentIndex = 0;
        flatten_KDTree(kdt.root_node_, h_array, currentIndex);

        // Transfer to device
        d_array = h_array;
        d_vAcc = h_vAcc;
        d_pct = h_pct;
    }

    void FlattenedKDTree::flatten_KDTree(const KDTree::Node* root, thrust::host_vector<ArrayNode>& array, size_t& currentIndex)
    {
        if (root == nullptr) return;

        size_t index = currentIndex++;
        array.resize(index + 1);

        if (root->child1 == nullptr && root->child2 == nullptr) {
            // Leaf node
            array[index].is_leaf = true;
            array[index].data.leaf.left = root->node_type.lr.left;
            array[index].data.leaf.right = root->node_type.lr.right;
        }
        else {
            // Non-leaf node
            array[index].is_leaf = false;
            array[index].data.nonleaf.divfeat = root->node_type.sub.divfeat;
            array[index].data.nonleaf.divlow = root->node_type.sub.divlow;
            array[index].data.nonleaf.divhigh = root->node_type.sub.divhigh;

            // Recursively flatten left and right child nodes
            size_t child1Index = currentIndex;
            flatten_KDTree(root->child1, array, currentIndex);
            array[index].data.nonleaf.child1 = child1Index;

            size_t child2Index = currentIndex;
            flatten_KDTree(root->child2, array, currentIndex);
            array[index].data.nonleaf.child2 = child2Index;
        }
    }

    __device__ __host__ float distance_squared(const Point3D p1, const Point3D p2)
    {
        float dx = p1.x - p2.x;
        float dy = p1.y - p2.y;
        float dz = p1.z - p2.z;
        return dx * dx + dy * dy + dz * dz;
    }

    __device__ __host__ void FlattenedKDTree::find_nearest_neighbor(const Point3D query, size_t index, float &best_dist, size_t &best_idx, int depth) const
    {
#ifdef  __CUDA_ARCH__
        if (index >= d_array.size()) return;
        const ArrayNode& node = d_array[index];
#else
        if (index >= h_array.size()) return;
        const ArrayNode& node = h_array[index]; 
#endif
        if (node.is_leaf)
        {
            // Leaf node: Check all points in the leaf node
            size_t left = node.data.leaf.left;
            size_t right = node.data.leaf.right;
            for (size_t i = left; i <= right; i++)
            {
#ifdef __CUDA_ARCH__
                float dist = distance_squared(query, d_pct[d_vAcc[i]]);
                if (dist < best_dist)
                {
                    best_dist = dist;
                    best_idx = d_vAcc[i];
                }
#else
                float dist = distance_squared(query, h_pct[h_vAcc[i]]);
                if (dist < best_dist)
                {
                    best_dist = dist;
                    best_idx = h_vAcc[i];
                }
#endif
            }
        }
        else
        {
            // Non-leaf node: Determine which child to search
            int axis = node.data.nonleaf.divfeat;
            float diff = query[axis] - node.data.nonleaf.divlow;

            // Choose the near and far child based on comparison
            size_t nearChild = diff < 0 ? node.data.nonleaf.child1 : node.data.nonleaf.child2;
            size_t farChild = diff < 0 ? node.data.nonleaf.child2 : node.data.nonleaf.child1;

            // Search near child
            find_nearest_neighbor(query, nearChild, best_dist, best_idx, depth + 1);

            // Search far child if needed
            if (diff * diff < best_dist)
            {
                find_nearest_neighbor(query, farChild, best_dist, best_idx, depth + 1);
            }
        }
    }

}